#include "hip/hip_runtime.h"
#include "../headers/matrix_ops.cuh"
#include "../headers/types"
#include <hip/hip_fp16.h>
#include <type_traits>

uint32 getMatrixId()
{
    static uint32 id = 0;
    return id++;
}

inline __device__ __host__ uint32 iDivUp(uint32 a, uint32 b) { return (a + b - 1) / b; }

constexpr uint32 BLOCK_SIZE_MM = 16;
constexpr uint32 BLOCK_SIZE = 32;

// AccumNative => internal accumulation is done in Tr, else 32bit(T) if sizeof(T) < 4 else 64bit(T)
template <uint32 TILE_SIZE_X = 16, bool AccumNative = false, typename Ta, typename Tb, typename Tc,
          typename Tr>
__global__ void tiled_madd_shmem(const Ta *__restrict__ A, uint32 aH, uint32 aW,
                                 const Tb *__restrict__ B, uint32 bW, const Tc *__restrict__ C,
                                 Tr *__restrict__ result)
{
    using SumType = typename std::conditional<AccumNative, Tr, typename AccumT<Tr>::type>::type;
    SumType sum{0};

    __shared__ float As[TILE_SIZE_X][TILE_SIZE_X + 1];
    __shared__ float Bs[TILE_SIZE_X][TILE_SIZE_X + 1];

    uint32 row = blockIdx.x * TILE_SIZE_X + threadIdx.x;
    uint32 col = blockIdx.y * TILE_SIZE_X + threadIdx.y;

    uint32 bH = aW;

#pragma unroll
    for (uint32 k = 0; k < iDivUp(aW, TILE_SIZE_X) * TILE_SIZE_X; k += TILE_SIZE_X)
    {
        uint32 aoffset = row * aW + k + threadIdx.y;
        uint32 boffset = (k + threadIdx.x) * bW + col;
        As[threadIdx.x][threadIdx.y] = (k + threadIdx.y < aW && row < aH) ? A[aoffset] : Ta(0);
        Bs[threadIdx.x][threadIdx.y] = (k + threadIdx.x < bH && col < bW) ? B[boffset] : Tb(0);
        __syncthreads();
#pragma unroll
        for (uint32 kk = 0; kk < TILE_SIZE_X; kk++)
        {
            sum += SumType(As[threadIdx.x][kk] * Bs[kk][threadIdx.y]);
        }
        __syncthreads();
    }
    if (row < aH && col < bW)
    {
        uint32 offset = row * bW + col;
        sum += SumType(C ? Tc(C[offset]) : Tc(0));
        result[offset] = sum;
    }
}

template <typename Ta, typename Tb, typename Tc, typename Tr>
__global__ void madd_kernel(const Ta *__restrict__ A, uint32 aH, uint32 aW,
                            const Tb *__restrict__ B, uint32 bW, const Tc *__restrict__ C,
                            Tr *__restrict__ result)
{
    uint32 i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32 j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < aH && j < bW)
    {
        Tr sum = 0;
#pragma unroll
        for (uint32 k = 0; k < aW; k++)
        {
            sum += A[i * aW + k] * B[k * bW + j];
        }
        result[i * bW + j] = sum + (C ? C[i * bW + j] : Tr(0));
    }
}

template <typename Tr, typename Ta, typename Tb, typename Tc>
void madd(Matrix<Tr> &result, const Matrix<Ta> &A, const Matrix<Tb> &B, const Matrix<Tc> *C)
{
    check_madd_sizes(result, A, B, C);

    if (result.numels() <= 1024 and false) // small matrices
    {
        // LOG("Using madd_kernel: ", result.numels());
        madd_kernel<Tr, Ta, Tb, Tc>
            <<<1, dim3(A.height, B.width)>>>(A.begin(), A.height, A.width, B.begin(), B.width,
                                             C ? C->begin() : nullptr, result.begin());
    }
    else if (result.width < BLOCK_SIZE_MM * 3 / 4) // skinny result matrix
    {
        dim3 blockDim(BLOCK_SIZE_MM, BLOCK_SIZE_MM);
        dim3 gridDim(iDivUp(A.height, BLOCK_SIZE_MM), iDivUp(B.width, BLOCK_SIZE_MM));
        tiled_madd_shmem<BLOCK_SIZE_MM, true, Tr, Ta, Tb, Tc>
            <<<gridDim, blockDim>>>(A.begin(), A.height, A.width, B.begin(), B.width,
                                    C ? C->begin() : nullptr, result.begin());
    }
    else
    {
        dim3 blockDim(BLOCK_SIZE_MM, BLOCK_SIZE_MM);
        dim3 gridDim(iDivUp(A.height, BLOCK_SIZE_MM), iDivUp(B.width, BLOCK_SIZE_MM));
        tiled_madd_shmem<BLOCK_SIZE_MM, false, Tr, Ta, Tb, Tc>
            <<<gridDim, blockDim>>>(A.begin(), A.height, A.width, B.begin(), B.width,
                                    C ? C->begin() : nullptr, result.begin());
    }
    cudaErrCheck(hipDeviceSynchronize());
}

template <typename T> Matrix<T> madd(const Matrix<T> &A, const Matrix<T> &B, const Matrix<T> *C)
{
    Matrix<T> result(A.height, B.width);
    fill(result, 0.0f);
    madd<T, T, T, T>(result, A, B, C);
    return result;
}

template <typename T>
__global__ void transpose_kernel(T *__restrict__ result, const T *__restrict__ A, uint32 height,
                                 uint32 width)
{
    __shared__ float tile[BLOCK_SIZE][BLOCK_SIZE + 1];
    uint32 x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    uint32 y = blockIdx.y * BLOCK_SIZE + threadIdx.y;

    if (x < width && y < height) tile[threadIdx.y][threadIdx.x] = A[y * width + x];

    __syncthreads();

    x = blockIdx.y * BLOCK_SIZE + threadIdx.x;
    y = blockIdx.x * BLOCK_SIZE + threadIdx.y;

    if (y < width && x < height) result[y * height + x] = tile[threadIdx.x][threadIdx.y];
}

template <typename T> Matrix<T> transpose(const Matrix<T> &A)
{
    uint32 max_dim = std::max(A.width, A.height);
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);

    dim3 gridDim(iDivUp(max_dim, BLOCK_SIZE), iDivUp(max_dim, BLOCK_SIZE));
    // LOG("A: ", A.height, "x", A.width, "blockDim: ", blockDim.x, "x", blockDim.y, " gridDim: ",
    // gridDim.x, "x", gridDim.y);
    Matrix<T> res(A.width, A.height);
    transpose_kernel<T><<<gridDim, blockDim>>>(res.begin(), A.begin(), A.height, A.width);
    cudaErrCheck(hipDeviceSynchronize());
    return res;
}

template void madd<float64, float64, float64>(Matrix<float64> &result, const Matrix<float64> &A,
                                              const Matrix<float64> &B, const Matrix<float64> *C);
template Matrix<float64> madd(const Matrix<float64> &A, const Matrix<float64> &B,
                              const Matrix<float64> *C);

template void madd<float32, float32, float32>(Matrix<float32> &result, const Matrix<float32> &A,
                                              const Matrix<float32> &B, const Matrix<float32> *C);
template Matrix<float32> madd(const Matrix<float32> &A, const Matrix<float32> &B,
                              const Matrix<float32> *C);

template void madd<float16, float16, float16, float16>(Matrix<float16> &result,
                                                       const Matrix<float16> &A,
                                                       const Matrix<float16> &B,
                                                       const Matrix<float16> *C);
template Matrix<half> madd(const Matrix<half> &A, const Matrix<half> &B, const Matrix<half> *C);

template Matrix<float32> transpose(const Matrix<float32> &A);