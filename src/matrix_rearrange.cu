#include "hip/hip_runtime.h"
#include "../headers/matrix_ops.cuh"

template <typename T, uint32 BLOCK_SIZE, typename Op>
__global__ void transpose_kernel(T *__restrict__ result, const T *__restrict__ A, uint32 height,
                                 uint32 width, Op op)
{
    __shared__ float32 tile[BLOCK_SIZE][BLOCK_SIZE + 1];
    uint32 x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    uint32 y = blockIdx.y * BLOCK_SIZE + threadIdx.y;

    if (x < width && y < height) tile[threadIdx.y][threadIdx.x] = A[y * width + x];

    __syncthreads();

    x = blockIdx.y * BLOCK_SIZE + threadIdx.x;
    y = blockIdx.x * BLOCK_SIZE + threadIdx.y;

    T out = op(tile[threadIdx.x][threadIdx.y]);

    if (y < width && x < height) result[y * height + x] = out;
}

template <typename T, typename Op>
void transpose(Matrix<T> &res, const Matrix<T> &A, Op op)
{
    if (A.height != res.width || A.width != res.height)
    {
        LOG(BOLD, RED, "Matrix dimensions do not match for transpose operation: ", A.shape_str,
            " -> ", res.shape_str);
        throw_rte_with_backtrace("Dimension mismatch for transpose");
    }

    if (A.width == 1 and std::is_same<Op, Identity<T>>::value)
    {
        fill(res, A.begin());
        return;
    }

    constexpr uint32 BLOCK_SIZE = 32;
    uint32 max_dim = std::max(A.width, A.height);
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);

    dim3 gridDim(iDivUp(max_dim, BLOCK_SIZE), iDivUp(max_dim, BLOCK_SIZE));
    transpose_kernel<T, BLOCK_SIZE, Op>
        <<<gridDim, blockDim>>>(res.begin(), A.begin(), A.height, A.width, op);
    cudaErrCheck(hipGetLastError());
}

template <typename T, typename Op = Identity<T>>
__global__ void concat_kernel(T *__restrict__ result, T **__restrict__ inputs, uint32 height,
                              uint32 width, Op op)
{
    uint32 x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32 y = blockIdx.y * blockDim.y + threadIdx.y;
    uint32 z = blockIdx.z;
    uint32 num_inputs = gridDim.z;

    if (x >= width || y >= height) return;
    uint32 out_offset =
        (width * num_inputs) * y + z * width + x;  // (larger width) * y + z * width + x
    result[out_offset] = op(inputs[z][y * width + x]);
}

static FloatT **concat_matrix_ptrs = nullptr;
static constexpr uint32 CONCAT_MAX = 128;
std::mutex concat_mtx;

template <typename T, typename Op>
void concat(Matrix<T> &res, const std::vector<Matrix<T> *> &inputs, Op op)
{
    std::lock_guard<std::mutex> lock(concat_mtx);
    if (concat_matrix_ptrs == nullptr)
    {
        cudaErrCheck(hipMallocManaged(&concat_matrix_ptrs, CONCAT_MAX * sizeof(FloatT *)));
    }

    if (inputs.size() > CONCAT_MAX)
    {
        throw_rte_with_backtrace("Number of matrices to concatenate exceeds ", CONCAT_MAX);
    }

    auto shape = inputs[0]->shape();
    for (uint32 i = 0; i < inputs.size(); i++)
    {
        if (inputs[i]->shape() != shape)
        {
            LOG(BOLD, RED,
                "Matrix shapes do not match for concatenation operation: ", inputs[i]->shape_str,
                " -> ", inputs[0]->shape_str);
            throw_rte_with_backtrace("Dimension mismatch for concatenation");
        }
        concat_matrix_ptrs[i] = inputs[i]->begin();
    }

    if (shape.first != res.height || shape.second * inputs.size() != res.width)
    {
        LOG(BOLD, RED, "Matrix dimensions do not match for concatenation operation: ", shape.first,
            "x", shape.second, " -> ", res.shape_str);
        throw_rte_with_backtrace("Dimension mismatch for concatenation");
    }

    dim3 blockDim(32, 32);
    dim3 gridDim(iDivUp(shape.second, 32), iDivUp(shape.first, 32), inputs.size());
    concat_kernel<T, Op>
        <<<gridDim, blockDim>>>(res.begin(), concat_matrix_ptrs, shape.first, shape.second, op);
}

template <typename T, typename Op>
__global__ void split_kernel(T **__restrict__ outputs, const T *__restrict__ input, uint32 height, uint32 width, Op op)
{
    uint32 x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32 y = blockIdx.y * blockDim.y + threadIdx.y;
    uint32 z = blockIdx.z;
    uint32 num_outputs = gridDim.z;

    if (x >= width || y >= height) return;
    uint32 in_offset =
        (width * num_outputs) * y + z * width + x;  // (larger width) * y + z * width + x
    outputs[z][y * width + x] = input[in_offset];
}

static FloatT **split_matrix_ptrs = nullptr;
static constexpr uint32 SPLIT_MAX = 128;
std::mutex split_mtx;

template <typename T, typename Op>
void split(std::vector<Matrix<T> *> &outputs, const Matrix<T> &res, Op op)
{
    std::lock_guard<std::mutex> lock(split_mtx);
    if (split_matrix_ptrs == nullptr)
    {
        cudaErrCheck(hipMallocManaged(&split_matrix_ptrs, SPLIT_MAX * sizeof(FloatT *)));
    }

    if (outputs.size() > SPLIT_MAX)
    {
        throw_rte_with_backtrace("Number of matrices to split exceeds ", SPLIT_MAX);
    }

    auto shape = outputs[0]->shape();
    for (uint32 i = 0; i < outputs.size(); i++)
    {
        if (outputs[i]->shape() != shape)
        {
            LOG(BOLD, RED,
                "Matrix shapes do not match for split operation: ", outputs[i]->shape_str, " -> ",
                outputs[0]->shape_str);
            throw_rte_with_backtrace("Dimension mismatch for split");
        }
        split_matrix_ptrs[i] = outputs[i]->begin();
    }

    if (shape.first != res.height || shape.second * outputs.size() != res.width)
    {
        LOG(BOLD, RED, "Matrix dimensions do not match for split operation: ", res.shape_str,
            " -> ", shape.first, "x", shape.second);
        throw_rte_with_backtrace("Dimension mismatch for split");
    }

    dim3 blockDim(32, 32);
    dim3 gridDim(iDivUp(shape.second, 32), iDivUp(shape.first, 32), outputs.size());
    split_kernel<T, Op>
        <<<gridDim, blockDim>>>(split_matrix_ptrs, res.begin(), shape.first, shape.second, op);
}

template void transpose(Matrix<FloatT> &res, const Matrix<FloatT> &A, Identity<FloatT>);

template void transpose<FloatT, Exp<FloatT>>(Matrix<FloatT> &, Matrix<FloatT> const &, Exp<FloatT>);

template void transpose<FloatT, Neg<FloatT>>(Matrix<FloatT> &, Matrix<FloatT> const &, Neg<FloatT>);

template void concat<FloatT, Identity<FloatT>>(
    Matrix<FloatT> &, std::vector<Matrix<FloatT> *, std::allocator<Matrix<FloatT> *>> const &,
    Identity<FloatT>);


template void split<FloatT, Identity<FloatT> >(std::vector<Matrix<FloatT>*, std::allocator<Matrix<FloatT>*> >&, Matrix<FloatT> const&, Identity<FloatT>);