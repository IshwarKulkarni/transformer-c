#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <type_traits>
#include "../headers/matrix_ops.cuh"
#include "../headers/types"

uint32 MatrixIds::alloced_byes = 0;
uint32 MatrixIds::id = 0;

template <typename Tc>
__device__ Tc getC(uint32 row, uint32 col, const Tc *C, uint32 cH, uint32 cW)
{
    if (C == nullptr) return Tc(0);
    uint32 y = cH > 1 ? row : 0;
    uint32 x = cW > 1 ? col : 0;
    return C[y * cW + x];
}

// AccumNative => internal accumulation is done in T, else 32bit(T) if sizeof(T)
// < 4 else 64bit(T)
template <uint32 TILE_SIZE_X = 16, bool AccumNative = false, typename T, typename PProcess>
__global__ void tiled_mmadd_shmem(T *__restrict__ result, const T *__restrict__ A, uint32 aH,
                                  uint32 aW, const T *__restrict__ B, uint32 bW,
                                  const T *__restrict__ C, uint32 cH, uint32 cW, PProcess pprocess)
{
    using SumType = typename std::conditional<AccumNative, T, typename AccumT<T>::type>::type;
    SumType sum{0};

    __shared__ T As[TILE_SIZE_X][TILE_SIZE_X + 1];
    __shared__ T Bs[TILE_SIZE_X][TILE_SIZE_X + 1];

    uint32 row = blockIdx.x * TILE_SIZE_X + threadIdx.x;
    uint32 col = blockIdx.y * TILE_SIZE_X + threadIdx.y;

    uint32 bH = aW;

#pragma unroll
    for (uint32 k = 0; k < iDivUp(aW, TILE_SIZE_X) * TILE_SIZE_X; k += TILE_SIZE_X)
    {
        uint32 aoffset = row * aW + k + threadIdx.y;
        uint32 boffset = (k + threadIdx.x) * bW + col;
        As[threadIdx.x][threadIdx.y] = (k + threadIdx.y < aW && row < aH) ? A[aoffset] : T(0);
        Bs[threadIdx.x][threadIdx.y] = (k + threadIdx.x < bH && col < bW) ? B[boffset] : T(0);
        __syncthreads();
#pragma unroll
        for (uint32 kk = 0; kk < TILE_SIZE_X; kk++)
        {
            sum += SumType(As[threadIdx.x][kk] * Bs[kk][threadIdx.y]);
        }
        __syncthreads();
    }

    if (row < aH && col < bW)
    {
        uint32 offset = row * bW + col;
        sum += getC(row, col, C, cH, cW);
        result[offset] = pprocess(sum);
    }
}

template <typename T, typename PProcess>
__global__ void mmadd_kernel(T *__restrict__ result, const T *__restrict__ A, uint32 aH, uint32 aW,
                             const T *__restrict__ B, uint32 bW, const T *__restrict__ C, uint32 cH,
                             uint32 cW, PProcess pprocess)
{
    uint32 i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32 j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < aH && j < bW)
    {
        T sum = 0;
#pragma unroll
        for (uint32 k = 0; k < aW; k++)
        {
            sum += A[i * aW + k] * B[k * bW + j];
        }

        result[i * bW + j] = pprocess(sum + getC(i, j, C, cH, cW));
    }
}

// A (h,w)  * B (w, 1)  + <C(h,1> -> result (h, 1)
// Assuming that B is column vector, if h > 1024, multiple calls to this kernel
// will be made with assuming h = 1024, followed by h=256  ...
template <typename T, uint32 BLOCK_X, typename PostProcess>
__global__ void mat_vector_mul_kernel(T *result, const T *A, const T *B, const T *C, uint32 height,
                                      uint32 width, uint32 offset = 0, bool addToresult = false,
                                      PostProcess pProcess = Identity<T>())
{
    uint32 x = threadIdx.x;
    uint32 y = blockIdx.x;
    __shared__ T As[BLOCK_X + 1];

    uint32 offset_x = x + offset;

    As[threadIdx.x] = (offset_x < width) ? T(A[offset_x + y * width] * B[offset_x]) : T(0);

    __syncthreads();

    T c = (C ? C[blockIdx.x] : T(0));
    T r = (addToresult ? result[blockIdx.x] : T(0));
#pragma unroll
    for (uint32 s = blockDim.x / 2; s > 32; s >>= 1)
    {
        if (x < s) As[x] += As[x + s];
        __syncthreads();
    }
    __syncthreads();
    volatile T *vAs = (volatile T *)As;
    if (x <= 32 and offset_x < width)
    {
        if (BLOCK_X >= 64) vAs[x] += vAs[x + 32];
        if (BLOCK_X >= 32) vAs[x] += vAs[x + 16];
        if (BLOCK_X >= 16) vAs[x] += vAs[x + 8];
        if (BLOCK_X >= 8) vAs[x] += vAs[x + 4];
        if (BLOCK_X >= 4) vAs[x] += vAs[x + 2];
        if (BLOCK_X >= 2) vAs[x] += vAs[x + 1];
    }
    __syncthreads();
    if (x == 0 and blockIdx.x < height)
    {
        result[blockIdx.x] = pProcess(vAs[0] + c + r);
    }
}

template <typename T, typename PostProcess>
__global__ void outer_product(T *result, const T *A, const T *B, const T *C, uint32 rheight,
                              uint32 rwidth, bool addToresult = false,
                              PostProcess pProcess = Identity<T>())
{
    uint32 x = threadIdx.x;
    uint32 y = blockIdx.x;

    if (x < rwidth and y < rheight)
    {
        T c = (C ? C[y * rwidth + x] : T(0));
        T r = (addToresult ? result[y * rwidth + x] : T(0));
        result[y * rwidth + x] = pProcess(T(A[y] * B[x]) + c + r);
    }
}

template <typename T, typename PostProcess>
void mvadd(Matrix<T> &result, const Matrix<T> &A, const Matrix<T> &B, const Matrix<T> *C,
           PostProcess pProcess)
{
    check_mmadd_sizes(result, A, B, C);

    if (B.height <= 16)
    {
        mat_vector_mul_kernel<T, 16><<<A.height, 16>>>(result.begin(), A.begin(), B.begin(),
                                                       C ? C->begin() : nullptr, A.height, A.width,
                                                       0, false, pProcess);
    }
    else if (B.height <= 32)
    {
        mat_vector_mul_kernel<T, 32><<<A.height, 32>>>(result.begin(), A.begin(), B.begin(),
                                                       C ? C->begin() : nullptr, A.height, A.width,
                                                       0, false, pProcess);
    }
    else if (B.height <= 64)
    {
        mat_vector_mul_kernel<T, 64><<<A.height, 64>>>(result.begin(), A.begin(), B.begin(),
                                                       C ? C->begin() : nullptr, A.height, A.width,
                                                       0, false, pProcess);
    }
    else if (B.height <= 128)
    {
        mat_vector_mul_kernel<T, 128><<<A.height, 128>>>(result.begin(), A.begin(), B.begin(),
                                                         C ? C->begin() : nullptr, A.height,
                                                         A.width, 0, false, pProcess);
    }
    else if (B.height <= 256)
    {
        mat_vector_mul_kernel<T, 256><<<A.height, 256>>>(result.begin(), A.begin(), B.begin(),
                                                         C ? C->begin() : nullptr, A.height,
                                                         A.width, 0, false, pProcess);
    }
    else if (B.height <= 512)
    {
        mat_vector_mul_kernel<T, 512><<<A.height, 512>>>(result.begin(), A.begin(), B.begin(),
                                                         C ? C->begin() : nullptr, A.height,
                                                         A.width, 0, false, pProcess);
    }
    else if (B.height <= 1024)
    {
        LOG("Using mat_vector_mul_kernel, A, B: ", A.shape_str, " ", B.shape_str);
        mat_vector_mul_kernel<T, 1024><<<A.height, 1024>>>(result.begin(), A.begin(), B.begin(),
                                                           C ? C->begin() : nullptr, A.height,
                                                           A.width, 0, false, pProcess);
    }
    else if (B.height > 1024)
    {
        constexpr int32 BLOCK_X = 1024;
        int32 offset = 0;
        for (; offset < B.height - BLOCK_X; offset += BLOCK_X)
        {
            mat_vector_mul_kernel<T, BLOCK_X, Identity<T>>
                <<<A.height, BLOCK_X>>>(  // only multi plication and addition, do not add C
                    result.begin(), A.begin(), B.begin(), nullptr, A.height, A.width, offset,
                    offset > 0);
        }

        mat_vector_mul_kernel<T, BLOCK_X><<<A.height, BLOCK_X>>>(  // add C and apply post process
            result.begin(), A.begin(), B.begin(), C ? C->begin() : nullptr, A.height, A.width,
            offset, offset > 0, pProcess);
    }
}

template <typename T, typename PProcess>
void mmadd(Matrix<T> &result, const Matrix<T> &A, const Matrix<T> &B, const Matrix<T> *C,
           PProcess pProcess)
{
    check_mmadd_sizes(result, A, B, C);

    if (A.width == 1 and
        B.width <= 1024)  // outer product (small enough to fit in one thread block)
    {
        outer_product<<<A.height, B.width>>>(result.begin(), A.begin(), B.begin(),
                                             (C ? C->begin() : nullptr), A.height, B.width, false,
                                             pProcess);
    }
    else if (B.width == 1)
    {
        mvadd<T>(result, A, B, C, pProcess);
    }
    else if (result.numels() <= 1024)  // small matrices
    {
        // LOG("Using mmadd_kernel: ", result.numels());
        mmadd_kernel<T><<<1, dim3(A.height, B.width)>>>(
            result.begin(), A.begin(), A.height, A.width, B.begin(), B.width,
            C ? C->begin() : nullptr, C ? C->height : 0, C ? C->width : 0, pProcess);
    }
    else if (A.height <= 1536)
    {
        constexpr uint32 BLOCK_SIZE_MM = 16;
        dim3 blockDim(BLOCK_SIZE_MM, BLOCK_SIZE_MM);
        dim3 gridDim(iDivUp(A.height, BLOCK_SIZE_MM), iDivUp(B.width, BLOCK_SIZE_MM));
        tiled_mmadd_shmem<BLOCK_SIZE_MM, false><<<gridDim, blockDim>>>(
            result.begin(), A.begin(), A.height, A.width, B.begin(), B.width,
            C ? C->begin() : nullptr, C ? C->height : 0, C ? C->width : 0, pProcess);
    }
    else
    {
        constexpr uint32 BLOCK_SIZE_MM = 32;
        dim3 blockDim(BLOCK_SIZE_MM, BLOCK_SIZE_MM);
        dim3 gridDim(iDivUp(A.height, BLOCK_SIZE_MM), iDivUp(B.width, BLOCK_SIZE_MM));
        tiled_mmadd_shmem<BLOCK_SIZE_MM, false><<<gridDim, blockDim>>>(
            result.begin(), A.begin(), A.height, A.width, B.begin(), B.width,
            C ? C->begin() : nullptr, C ? C->height : 0, C ? C->width : 0, pProcess);
    }
    cudaErrCheck(hipGetLastError());
}

template <typename T, uint32 BLOCK_SIZE, typename Op>
__global__ void transpose_kernel(T *__restrict__ result, const T *__restrict__ A, uint32 height,
                                 uint32 width, Op op)
{
    __shared__ float32 tile[BLOCK_SIZE][BLOCK_SIZE + 1];
    uint32 x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    uint32 y = blockIdx.y * BLOCK_SIZE + threadIdx.y;

    if (x < width && y < height) tile[threadIdx.y][threadIdx.x] = A[y * width + x];

    __syncthreads();

    x = blockIdx.y * BLOCK_SIZE + threadIdx.x;
    y = blockIdx.x * BLOCK_SIZE + threadIdx.y;

    T out = op(tile[threadIdx.x][threadIdx.y]);

    if (y < width && x < height) result[y * height + x] = out;
}

template <typename T, typename Op>
void transpose(Matrix<T> &res, const Matrix<T> &A, Op op)
{
    if (A.height != res.width || A.width != res.height)
    {
        LOG(BOLD, RED, "Matrix dimensions do not match for transpose operation: ", A.shape_str,
            " -> ", res.shape_str);
        throw_rte_with_backtrace("Dimension mismatch for transpose");
    }

    if (A.width == 1 and std::is_same<Op, Identity<T>>::value)
    {
        fill(res, A.begin());
        return;
    }

    constexpr uint32 BLOCK_SIZE = 32;
    uint32 max_dim = std::max(A.width, A.height);
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);

    dim3 gridDim(iDivUp(max_dim, BLOCK_SIZE), iDivUp(max_dim, BLOCK_SIZE));
    transpose_kernel<T, BLOCK_SIZE, Op>
        <<<gridDim, blockDim>>>(res.begin(), A.begin(), A.height, A.width, op);
    cudaErrCheck(hipGetLastError());
}

template void mmadd<FloatT, Sigmoid<FloatT>::SigmoidF>(Matrix<FloatT> &, Matrix<FloatT> const &,
                                                       Matrix<FloatT> const &,
                                                       Matrix<FloatT> const *,
                                                       Sigmoid<FloatT>::SigmoidF);
template void mmadd<FloatT, Identity<FloatT>>(Matrix<FloatT> &, Matrix<FloatT> const &,
                                              Matrix<FloatT> const &, Matrix<FloatT> const *,
                                              Identity<FloatT>);
template void transpose(Matrix<FloatT> &res, const Matrix<FloatT> &A, Identity<FloatT>);

template void transpose<FloatT, Exp<FloatT>>(Matrix<FloatT> &, Matrix<FloatT> const &, Exp<FloatT>);

template void transpose<FloatT, Neg<FloatT>>(Matrix<FloatT> &, Matrix<FloatT> const &, Neg<FloatT>);

template void mmadd<FloatT, Relu<FloatT>::ReluF>(Matrix<FloatT> &, Matrix<FloatT> const &,
                                                 Matrix<FloatT> const &, Matrix<FloatT> const *,
                                                 Relu<FloatT>::ReluF);

template void mmadd<FloatT, TanH<FloatT>::TanhF>(Matrix<FloatT> &, Matrix<FloatT> const &,
                                                 Matrix<FloatT> const &, Matrix<FloatT> const *,
                                                 TanH<FloatT>::TanhF);

template void mmadd<FloatT, DividebBy<FloatT>>(Matrix<FloatT> &, Matrix<FloatT> const &,
                                               Matrix<FloatT> const &, Matrix<FloatT> const *,
                                               DividebBy<FloatT>);

template void mmadd<FloatT, Neg<FloatT>>(Matrix<FloatT> &, Matrix<FloatT> const &,
                                         Matrix<FloatT> const &, Matrix<FloatT> const *,
                                         Neg<FloatT>);